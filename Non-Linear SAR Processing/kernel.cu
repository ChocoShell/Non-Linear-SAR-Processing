#include "hip/hip_runtime.h"
/*
Programming the algorithms in Non-Linear SAR Processing paper
1. Choose a transmitted radar signal
4. Simulate measured SAR Data signal in fast and slow time (signal frequency, plane)
5. Convolution or FFT and IFFT for equation 6
6. Code equation 12
7. Section 3.5 mx, my = xi, yi = location of reflector

//Further Stuff
Read Section 4 and 5 as separate paper

//New Plan
0. Figure out how to plot 3d graphs in C++/CUDA
1. Randomly Generate Map (X v Y with amplitudes -> 250 x 100, x = 1000:2:1500, y = 100:200)
2. Plot Map on Figure 1
3. Generate Plane Function length 100 -> x in range [0,50] y in range [50,250]
4. Plot Plane on Figure 1
5. Generate Radar Signal (Simple Chirp)
6. Plot Radar Signal on Figure 2
7. Simulate SAR Data -> Get delay and attenuation(magnitude) from map(x,y), add gaussian noise to return signal
                     -> Do this 100 times 
8. Get sm(x,y,u) = sm(t,u) = s(x, y, u) * p*(-t) -> Same length use as function indices not matching delays
9. Integrate sm(x,y,u) over u to get f(x,y)
10. Plot f(x,y) compare with the first map

// Help from:
http://nehe.gamedev.net/tutorial/your_first_polygon/13002/
http://www.wikihow.com/Make-a-Cube-in-OpenGL

New New Plan
Write own convolution function
plot sM in matlab

*/
// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

#include <iostream>
#include <fstream>
#include <cstdlib>
#include <list>

#ifdef _WIN32
#  define WINDOWS_LEAN_AND_MEAN
#  define NOMINMAX
#  include <windows.h>
#endif

// OpenGL Graphics includes
#include <GL/glew.h>
#include <GL/freeglut.h>

// CUDA includes
#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>
#include ""

// Utilities and timing functions
#include <helper_functions.h>    // includes hip/hip_runtime.h and hip/hip_runtime_api.h
#include <timer.h>               // timing functions

// CUDA helper functions
#include <hip/hip_runtime_api.h>         // helper functions for CUDA error check
#include <helper_cuda_gl.h>      // helper functions for CUDA/GL interop

// CUDA Libraries
#include <hipfft/hipfft.h>
#include <hipfft/hipfftw.h>
#include <hiprand/hiprand_kernel.h>

#include <hip/hip_vector_types.h>

#include <array>

using namespace std;

#define BLOCK_SIZE 32

#define PI 3.1415926535

const long int spd_of_light = 299792458;

void split_line(string& line, string delim, list<string>& values)
{
    size_t pos = 0;
    while ((pos = line.find(delim, (pos + 1))) != string::npos) {
        string p = line.substr(0, pos);
        values.push_back(p);
        line = line.substr(pos + 1);
    }

    if (!line.empty()) {
        values.push_back(line);
    }
}

__global__ void flattenKernel(hipDoubleComplex *matrix_signal, hipDoubleComplex *out_signal, const int width, const int batch)
{
	hipDoubleComplex ducks;
	unsigned int r = (blockIdx.x * blockDim.x) + threadIdx.x;
	if (width > r)
	{
		for (int i = 0; i < batch; i ++)
		{
			ducks.x += matrix_signal[i * batch + r].x;
			ducks.y += matrix_signal[i * batch + r].y;
		}
		out_signal[r] = ducks;
	}
}

__global__ void conv_mat_vec_kernel(hipDoubleComplex *matrix, hipDoubleComplex *vector, hipDoubleComplex *out, const unsigned int width, const unsigned int batch)
{
	int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int maxLen = 2*width - 1;
    float matX, matY, vecX, vecY;
	unsigned int start, end;
	hipDoubleComplex sum;
	
    if (batch -1 < row || maxLen - 1 < col) return;

	sum.x = 0;
	sum.y = 0;
	
	if (col < width)
	{
		start = 0; // greater or equal to this
		end   = col +1; // less than this
	}
	else
	{
		start = col - width +1;
		end   = width; //less than this
	}

    //start and end act as Tau in the convolution equation
	for(start; start < end; start++)
	{
        matX = matrix[start].x;
    	matY = matrix[start].y;
	    vecX = vector[col-start].x;
	    vecY = vector[col-start].y;
        sum.x = matX*vecX - matY*vecY;
		sum.y = matX*vecY + matY*vecX;
	}
	out[row*maxLen + col] = sum;	
}

void convolveWithCuda(hipDoubleComplex *unknown_signal_block, hipDoubleComplex *template_signal, hipDoubleComplex *hOut, const int width, const int batch)
{	
	hipDoubleComplex *data, *temp, *out, curr;
    int x,y;
    int tots = 2*width - 1;

	// FFT of return signal matrix
	hipMalloc((void**)&data, sizeof(hipDoubleComplex)*width*batch);
	if (hipGetLastError() != hipSuccess)
	{
		fprintf(stderr, "Cuda error: Failed to allocate data\n");
		return;
	}

	hipMalloc((void**)&out, sizeof(hipDoubleComplex)*(2*width - 1)*batch);
	if (hipGetLastError() != hipSuccess)
	{
		fprintf(stderr, "Cuda error: Failed to allocate data\n");
		return;
	}

	hipMalloc((void**)&temp, sizeof(hipDoubleComplex)*width);
	if (hipGetLastError() != hipSuccess)
	{
		fprintf(stderr, "Cuda error: Failed to allocate temp\n");
		return;
	}

	hipMemcpy(data, unknown_signal_block, sizeof(hipDoubleComplex)*batch*width, hipMemcpyHostToDevice);
	hipMemcpy(temp, template_signal,      sizeof(hipDoubleComplex)*width,       hipMemcpyHostToDevice);
    
    dim3 threadsPerBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 numOfBlocks((2*width-1)/threadsPerBlock.x + 1, batch/threadsPerBlock.y + 1);

    conv_mat_vec_kernel<<< numOfBlocks, threadsPerBlock >>>(data, temp, out, width, batch);
    
    hipDeviceSynchronize();
    
    int err = hipGetLastError();

	hipMemcpy(hOut, out, sizeof(hipDoubleComplex)*(2*width-1)*batch, hipMemcpyDeviceToHost);
    
    if (err != hipSuccess)
	{
		fprintf(stderr, "Cuda error: Failed to Synchronize\n");
        cout << err << endl;
		return;
	}

    for(x = 0; x < batch; x++)
    {
        for(y = 0; y < tots; y++)
        {
            curr = hOut[x* tots + y];
            printf("%g + (%gi), ", hipCreal(curr), hipCimag(curr));
        }
        cout << endl;
    }

	hipFree(data);
	hipFree(temp);
	hipFree(out);
	return;
}

int main()
{
	//new code
	hipDeviceReset();
	hipSetDevice(0);

	// Get all data
	ifstream fastTimeFilter ("fastTimeFilter.csv");
	ifstream imagsRaw ("imagsRaw.csv");
	ifstream realsRaw ("realsRaw.csv");
	// End of data read

	float d, i;
	int count = 437;
	hipDoubleComplex *sRaw, *signal, *hOut;
	string value;
	hOut   = (hipDoubleComplex *)malloc(sizeof(hipDoubleComplex)*(438*2 - 1)*160);
	signal = (hipDoubleComplex *)malloc(sizeof(hipDoubleComplex)*438);
	sRaw   = (hipDoubleComplex *)malloc(sizeof(hipDoubleComplex)*438*160);
    hipDoubleComplex curr;
    int width = 438;
    int batch = 160;
    int x,y;
    int tots = 2*width - 1;

    list<string> values;

    while ( fastTimeFilter.good() )
    {
        getline ( fastTimeFilter, value, ',' ); // read a string until next comma: http://www.cplusplus.com/reference/string/getline/
        if (value.find('\n') != string::npos) {
            split_line(value, "\n", values);
        } else {
            values.push_back(value);
        }
    }

    list<string>::const_iterator it = values.begin();
    for (it = values.begin(); it != values.end(); it++) {
        string tmp = *it;
        d = stof(tmp.c_str(), NULL);
		it++;
		tmp = *it;
		i = stof(tmp.c_str(), NULL);
		signal[count].x = d;
		signal[count].y = -1 * i;
		count--;
        //cout << "Double val: " << right << showpoint << d << endl;
    }
	
	string value1;
    list<string> values1;
    while ( imagsRaw.good() )
    {
        getline ( imagsRaw, value1, ',' ); // read a string until next comma: http://www.cplusplus.com/reference/string/getline/
        if (value1.find('\n') != string::npos) {
            split_line(value1, "\n", values1);
        } else {
            values1.push_back(value1);
        }
    }

    it = values1.begin();
    count = 0;
    for (it = values1.begin(); it != values1.end(); it++) {
        string tmp = *it;
        d = stof(tmp.c_str(), NULL);
        sRaw[count].y = d;
        count++;
        //cout << "Double val: " << right << showpoint << d << endl;
    }

    string value2;
    list<string> values2;
    while ( realsRaw.good() )
    {
        getline ( realsRaw, value2, ',' ); // read a string until next comma: http://www.cplusplus.com/reference/string/getline/
        if (value2.find('\n') != string::npos) {
            split_line(value2, "\n", values2);
        } else {
            values2.push_back(value2);
        }
    }

    it = values2.begin();
    count = 0;
    for (it = values2.begin(); it != values2.end(); it++) {
        string tmp = *it;
        d = stof(tmp.c_str(), NULL);
		sRaw[count].x = d;
		count++;
        //cout << "Double val: " << right << showpoint << d << endl;
    }
	//Done Reading in values from files.

	convolveWithCuda(sRaw, signal, hOut, 438, 160);
	//hOut is sM after convolution of raw data with p signal
    //x -> 1-382, y-> 1-266
    // Loop through both x and y for each u to get an image at a specific u, then add all the images up (combining them by their u)
    // 
    free(hOut);
	free(signal);
	free(sRaw);
	hipDeviceReset();
	return 0;
}