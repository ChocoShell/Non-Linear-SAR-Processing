#include "hip/hip_runtime.h"
/*
Programming the algorithms in Non-Linear SAR Processing paper
1. Choose a transmitted radar signal
4. Simulate measured SAR Data signal in fast and slow time (signal frequency, plane)
5. Convolution or FFT and IFFT for equation 6
6. Code equation 12
7. Section 3.5 mx, my = xi, yi = location of reflector

//Further Stuff
Read Section 4 and 5 as separate paper

//New Plan
0. Figure out how to plot 3d graphs in C++/CUDA
1. Randomly Generate Map (X v Y with amplitudes -> 250 x 100, x = 1000:2:1500, y = 100:200)
2. Plot Map on Figure 1
3. Generate Plane Function length 100 -> x in range [0,50] y in range [50,250]
4. Plot Plane on Figure 1
5. Generate Radar Signal (Simple Chirp)
6. Plot Radar Signal on Figure 2
7. Simulate SAR Data -> Get delay and attenuation(magnitude) from map(x,y), add gaussian noise to return signal
                     -> Do this 100 times 
8. Get sm(x,y,u) = sm(t,u) = s(x, y, u) * p*(-t) -> Same length use as function indices not matching delays
9. Integrate sm(x,y,u) over u to get f(x,y)
10. Plot f(x,y) compare with the first map

// Help from:
http://nehe.gamedev.net/tutorial/your_first_polygon/13002/
http://www.wikihow.com/Make-a-Cube-in-OpenGL

New New Plan
Write own convolution function
plot sM in matlab

*/
// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>

#include <iostream>
#include <fstream>
#include <cstdlib>
#include <list>

#ifdef _WIN32
#  define WINDOWS_LEAN_AND_MEAN
#  define NOMINMAX
#  include <windows.h>
#endif

// OpenGL Graphics includes
#include <GL/glew.h>
#include <GL/freeglut.h>

// CUDA includes
#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>
#include ""

// Utilities and timing functions
#include <helper_functions.h>    // includes hip/hip_runtime.h and hip/hip_runtime_api.h
#include <timer.h>               // timing functions

// CUDA helper functions
#include <hip/hip_runtime_api.h>         // helper functions for CUDA error check

// CUDA Libraries
#include <hipfft/hipfft.h>
#include <hipfft/hipfftw.h>
#include <hiprand/hiprand_kernel.h>
#include <math_functions.h>

#include <hip/hip_vector_types.h>

#include <array>

using namespace std;

#define BLOCK_SIZE 32

#define PI 3.1415926535

const long int spd_of_light = 299792458;

void split_line(string& line, string delim, list<string>& values)
{
    size_t pos = 0;
    while ((pos = line.find(delim, (pos + 1))) != string::npos) {
        string p = line.substr(0, pos);
        values.push_back(p);
        line = line.substr(pos + 1);
    }

    if (!line.empty()) {
        values.push_back(line);
    }
}

//kernel functions
__global__ void square_kernel(hipComplex *d_vector, hipComplex *d_out, const unsigned int length, const unsigned int width)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row >= length || col >= width) {return;}

    d_out[width*row + col] = hipCmulf(d_vector[width*row + col], d_vector[width*row + col]);
}
__global__ void sqrt_abs_kernel(hipComplex *d_in, hipComplex *d_out, const unsigned int length, const unsigned int width)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row >= length || col >= width) {return;}

    d_out[width*row + col].x = rsqrtf(hipCabsf(d_in[width*row + col]));
    d_out[width*row + col].y = 0;
}
__global__ void real_to_imag_kernel(hipComplex *d_in, hipComplex *d_out, const unsigned int length, const unsigned int width)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row >= length || col >= width) {return;}
    
    d_out[width*row + col].x = 0;
    d_out[width*row + col].y = d_in[width*row + col].x;
}
__global__ void vec_vec_mult_kernel(hipComplex *d_vec1, hipComplex *d_vec2, hipComplex *d_out, const unsigned int length, const unsigned int width)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    int newRow, ind;

    if (row*BLOCK_SIZE >= length || col >= width) {return;}

    for (int i = 0; i < BLOCK_SIZE; i++)
    {
        newRow = (BLOCK_SIZE*row) + i;
        if(newRow < length)
        {
            ind = col + width*newRow;
            d_out[ind] = hipCmulf(d_vec1[ind], d_vec2[ind]);
        }
    }
}
__global__ void vec_vec_mat_kernel(hipComplex *d_vec1, hipComplex *d_vec2, hipComplex *d_out, const unsigned int len_1, const unsigned int len_2)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row >= len_1 || col >= len_2) {return;}

    d_out[len_2*row + col] = hipCmulf(d_vec1[row], d_vec2[col]);
}
__global__ void sca_vec_add_kernel(const double K, hipComplex *d_vector, const unsigned length, const unsigned int width, const double M)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row >= length || col >= width) {return;}

    d_vector[width*row + col].x = K + M*d_vector[width*row + col].x;
    d_vector[width*row + col].y = M*d_vector[width*row + col].y;
}
__global__ void sca_vec_mult_kernel(const double K, hipComplex *d_vector, const unsigned int length, const unsigned int width)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    int newRow;

    if (row*BLOCK_SIZE >= length || col >= width) {return;}

    for (int i = 0; i < BLOCK_SIZE; i++)
    {
        newRow = (BLOCK_SIZE*row) + i;
        if(newRow < length)
        {
            d_vector[col + width*newRow].x *= K;
            d_vector[col + width*newRow].y *= K;
        }
    }
}
__global__ void transpose_kernel(hipComplex *d_matrix, hipComplex *d_out, const unsigned int length, const unsigned int width)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    
    int newRow;

    if (row*BLOCK_SIZE >= length || col >= width) {return;}

    for (int i = 0; i < BLOCK_SIZE; i++)
    {
        newRow = (BLOCK_SIZE*row) + i;
        if(newRow < length)
        {
            d_out[length*col + newRow] = d_matrix[col + width*newRow];
        }
    }
    return;
}
__global__ void fftshift_kernel(hipComplex *d_signal, hipComplex *d_out, const unsigned int width, const unsigned int batch, const int dim)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    
    if ((row >= batch) || (col >= width))
        return;

    int newcol = col;
    int newrow = row;

    if (dim != 1) 
        newcol = (col + width/2 +3) % width;

    if (dim != 2) 
        newrow = (row + batch/2 +3) % batch;
    
    d_out[newcol + newrow*width] = d_signal[col + row*width];
}
__global__ void map_kernel(hipComplex *s_M, hipComplex *out, const unsigned int width, const unsigned int batch, const unsigned int max_x, const unsigned int max_y)
{   /*s_M is the fast-time matched filtered SAR signal
     *Because this signal is in discrete time and uses indices 
     *instead of values, we have to modify the time delay
     *function to fit between the range 1 and the width of s_M
    */

    // We will run a total of max_x * max_y threads,
    // looping through all possible slow time locations
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if(max_y - 1 < row || max_x - 1 < col) return;

    int u;
    int delay;
    hipComplex val;
    val.x= 0;
    val.y =0;
    int uNormal = max_y/batch;
    for(u = 0; u < batch; u++)
    {   // The number 1.8821 is width/max(s_m), this will be changed later
        // It is used to evenly distribute the magnitudes of the map
        delay = lround(hypot( (double) row, (double) (col - u*uNormal))*0.01281);
        val.x += s_M[u*width + delay].x;
        val.y += s_M[u*width + delay].y;        
    }
    out[row*max_x + col] = val;
    return;
}
__global__ void mat_vec_mult_kernel(hipComplex *matrix, hipComplex *vector, hipComplex *out, const unsigned int width, const unsigned int batch)
{
	int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row >= batch || col >= width) {return;}

    out[row*width + col] = hipCmulf(matrix[row*width + col], vector[col]);
}

// kernel helpers
void square(hipComplex *h_vector, hipComplex *h_out, const unsigned int length, const unsigned int width)
{
    hipComplex *d_vector, *d_out;

    hipMalloc((void**)&d_vector, sizeof(hipComplex)*width*length);
    if (hipGetLastError() != hipSuccess)
	{
		fprintf(stderr, "Cuda error: Failed to allocate memory for matrix\n");
		return;
	}

    hipMalloc((void**)&d_out, sizeof(hipComplex)*width*length);
    if (hipGetLastError() != hipSuccess)
	{
		fprintf(stderr, "Cuda error: Failed to allocate memory for matrix\n");
		return;
	}

    hipMemcpy(d_vector, h_vector, sizeof(hipComplex)*width*length,
               hipMemcpyHostToDevice);
    if (hipGetLastError() != hipSuccess)
	{
		fprintf(stderr, "Cuda error: Memcpy to device failed\n");
		return;
	}

    dim3 threadsPerBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 numOfBlocks(width/threadsPerBlock.x + 1, length/threadsPerBlock.y + 1);

    square_kernel<<<numOfBlocks, threadsPerBlock>>>(d_vector, d_vector, length, width);

    hipMemcpy(h_vector, d_out, sizeof(hipComplex)*width*length,
               hipMemcpyDeviceToHost);
    if (hipGetLastError() != hipSuccess)
	{
		fprintf(stderr, "Cuda error: Memcpy to host failed\n");
		return;
	}

    hipFree(d_out);
    hipFree(d_vector);
}
void sqrt_abs(hipComplex *h_in, hipComplex *h_out, const unsigned int length, const unsigned int width)
{
    hipComplex *d_in, *d_out;

    hipMalloc((void**)&d_in, sizeof(hipComplex)*width*length);
    if (hipGetLastError() != hipSuccess)
	{
		fprintf(stderr, "Cuda error: Failed to allocate memory for matrix\n");
		return;
	}

    hipMalloc((void**)&d_out, sizeof(hipComplex)*width*length);
    if (hipGetLastError() != hipSuccess)
	{
		fprintf(stderr, "Cuda error: Failed to allocate memory for matrix\n");
		return;
	}

    hipMemcpy(d_in, h_in, sizeof(hipComplex)*width*length,
               hipMemcpyHostToDevice);
    if (hipGetLastError() != hipSuccess)
	{
		fprintf(stderr, "Cuda error: Memcpy to device failed\n");
		return;
	}

    dim3 threadsPerBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 numOfBlocks(width/threadsPerBlock.x + 1, length/threadsPerBlock.y + 1);

    sqrt_abs_kernel<<<numOfBlocks, threadsPerBlock>>>(d_in, d_out, length, width);

    hipMemcpy(h_out, d_out, sizeof(hipComplex)*width*length,
               hipMemcpyDeviceToHost);
    if (hipGetLastError() != hipSuccess)
	{
		fprintf(stderr, "Cuda error: Memcpy to host failed\n");
		return;
	}

    hipFree(d_in);
    hipFree(d_out);
}
void real_to_imag(hipComplex *h_in, hipComplex *h_out, const unsigned int length, const unsigned int width)
{
    hipComplex *d_in, *d_out;

    hipMalloc((void**)&d_in, sizeof(hipComplex)*width*length);
    if (hipGetLastError() != hipSuccess)
	{
		fprintf(stderr, "Cuda error: Failed to allocate memory for matrix\n");
		return;
	}

    hipMalloc((void**)&d_out, sizeof(hipComplex)*width*length);
    if (hipGetLastError() != hipSuccess)
	{
		fprintf(stderr, "Cuda error: Failed to allocate memory for matrix\n");
		return;
	}

    hipMemcpy(d_in, h_in, sizeof(hipComplex)*width*length,
               hipMemcpyHostToDevice);
    if (hipGetLastError() != hipSuccess)
	{
		fprintf(stderr, "Cuda error: Memcpy to device failed\n");
		return;
	}

    dim3 threadsPerBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 numOfBlocks(width/threadsPerBlock.x + 1, length/threadsPerBlock.y + 1);

    real_to_imag_kernel<<<numOfBlocks, threadsPerBlock>>>(d_in, d_out, length, width);

    hipMemcpy(h_out, d_out, sizeof(hipComplex)*width*length,
               hipMemcpyDeviceToHost);
    if (hipGetLastError() != hipSuccess)
	{
		fprintf(stderr, "Cuda error: Memcpy to host failed\n");
		return;
	}

    hipFree(d_in);
    hipFree(d_out);
}
void vec_vec_mult(hipComplex *h_vec1, hipComplex *h_vec2, const unsigned int length, const unsigned int width)
{
    //Element wise multiplication of 2 vectors, output is placed in h_vec1
    hipComplex *d_vec1, *d_vec2, *d_out;

    hipMalloc((void**)&d_vec1, sizeof(hipComplex)*width*length);
    if (hipGetLastError() != hipSuccess)
	{
		fprintf(stderr, "Cuda error: Failed to allocate memory for matrix\n");
		return;
	}

    hipMalloc((void**)&d_vec2, sizeof(hipComplex)*width*length);
    if (hipGetLastError() != hipSuccess)
	{
		fprintf(stderr, "Cuda error: Failed to allocate memory for matrix\n");
		return;
	}

    hipMalloc((void**)&d_out, sizeof(hipComplex)*width*length);
    if (hipGetLastError() != hipSuccess)
	{
		fprintf(stderr, "Cuda error: Failed to allocate memory for matrix\n");
		return;
	}

    //Copying vectors onto device
    hipMemcpy(d_vec1, h_vec1, sizeof(hipComplex)*width*length,
               hipMemcpyHostToDevice);
    if (hipGetLastError() != hipSuccess)
	{
		fprintf(stderr, "Cuda error: Memcpy to device failed\n");
		return;
	}

    hipMemcpy(d_vec2, h_vec2, sizeof(hipComplex)*width*length,
               hipMemcpyHostToDevice);
    if (hipGetLastError() != hipSuccess)
	{
		fprintf(stderr, "Cuda error: Memcpy to device failed\n");
		return;
	}

    dim3 threadsPerBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 numOfBlocks(width/(threadsPerBlock.x + BLOCK_SIZE) + 1, length/threadsPerBlock.y + 1);

    vec_vec_mult_kernel<<<numOfBlocks, threadsPerBlock>>>(d_vec1, d_vec2, d_out, length, width);

    hipMemcpy(h_vec1, d_out, sizeof(hipComplex)*width*length,
               hipMemcpyDeviceToHost);
    if (hipGetLastError() != hipSuccess)
	{
		fprintf(stderr, "Cuda error: Memcpy to host failed\n");
		return;
	}

    hipFree(d_vec1);
    hipFree(d_vec2);
    hipFree(d_out);
}
void vec_vec_mat(hipComplex *h_vec1, hipComplex *h_vec2, hipComplex *h_out, const unsigned int len_1, const unsigned int len_2)
{
    hipComplex *d_vec1, *d_vec2, *d_out;

    hipMalloc((void**)&d_vec1, sizeof(hipComplex)*len_1);
    if (hipGetLastError() != hipSuccess)
	{
		fprintf(stderr, "Cuda error: Failed to allocate memory for matrix\n");
		return;
	}

    hipMalloc((void**)&d_vec2, sizeof(hipComplex)*len_2);
    if (hipGetLastError() != hipSuccess)
	{
		fprintf(stderr, "Cuda error: Failed to allocate memory for matrix\n");
		return;
	}

    hipMalloc((void**)&d_out, sizeof(hipComplex)*len_1*len_2);
    if (hipGetLastError() != hipSuccess)
	{
		fprintf(stderr, "Cuda error: Failed to allocate memory for matrix\n");
		return;
	}

    //Copying vectors onto device
    hipMemcpy(d_vec1, h_vec1, sizeof(hipComplex)*len_1,
               hipMemcpyHostToDevice);
    if (hipGetLastError() != hipSuccess)
	{
		fprintf(stderr, "Cuda error: Memcpy to device failed\n");
		return;
	}

    hipMemcpy(d_vec2, h_vec2, sizeof(hipComplex)*len_2,
               hipMemcpyHostToDevice);
    if (hipGetLastError() != hipSuccess)
	{
		fprintf(stderr, "Cuda error: Memcpy to device failed\n");
		return;
	}

    dim3 threadsPerBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 numOfBlocks(len_2/threadsPerBlock.x + 1, len_1/threadsPerBlock.y + 1);

    vec_vec_mat_kernel<<<numOfBlocks, threadsPerBlock>>>(d_vec1, d_vec2, d_out, len_1, len_2);

    hipMemcpy(h_out, d_out, sizeof(hipComplex)*len_2*len_1,
               hipMemcpyDeviceToHost);
    if (hipGetLastError() != hipSuccess)
	{
		fprintf(stderr, "Cuda error: Memcpy to device failed\n");
		return;
	}
    
    hipFree(d_out);
    hipFree(d_vec1);
    hipFree(d_vec2);
}
void sca_vec_add(const double K, hipComplex *h_vector, const unsigned length, const unsigned int width, const double M)
{
    hipComplex *d_vector;

    hipMalloc((void**)&d_vector, sizeof(hipComplex)*width*length);
    if (hipGetLastError() != hipSuccess)
	{
		fprintf(stderr, "Cuda error: Failed to allocate memory for matrix\n");
		return;
	}

    hipMemcpy(d_vector, h_vector, sizeof(hipComplex)*width*length,
               hipMemcpyHostToDevice);
    if (hipGetLastError() != hipSuccess)
	{
		fprintf(stderr, "Cuda error: Memcpy to device failed\n");
		return;
	}

    dim3 threadsPerBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 numOfBlocks(width/threadsPerBlock.x + 1, length/threadsPerBlock.y + 1);

    sca_vec_add_kernel<<<numOfBlocks, threadsPerBlock>>>(K, d_vector, length, width, M);

    hipMemcpy(h_vector, d_vector, sizeof(hipComplex)*width*length,
               hipMemcpyDeviceToHost);
    if (hipGetLastError() != hipSuccess)
	{
		fprintf(stderr, "Cuda error: Memcpy to device failed\n");
		return;
	}

    hipFree(d_vector);
}
void sca_vec_mult(const double K, hipComplex *h_vector, const unsigned int length, const unsigned int width)
{
    hipComplex *d_vector;

    hipMalloc((void**)&d_vector, sizeof(hipComplex)*width*length);
    if (hipGetLastError() != hipSuccess)
	{
		fprintf(stderr, "Cuda error: Failed to allocate memory for matrix\n");
		return;
	}

    //Copying vector onto device
    hipMemcpy(d_vector, h_vector, sizeof(hipComplex)*width*length,
               hipMemcpyHostToDevice);
    if (hipGetLastError() != hipSuccess)
	{
		fprintf(stderr, "Cuda error: Memcpy to device failed\n");
		return;
	}

    dim3 threadsPerBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 numOfBlocks(width/(threadsPerBlock.x + BLOCK_SIZE) + 1, length/threadsPerBlock.y + 1);

    sca_vec_mult_kernel<<<numOfBlocks, threadsPerBlock>>>(K, d_vector, length, width);

    hipMemcpy(h_vector, d_vector, sizeof(hipComplex)*width*length,
               hipMemcpyDeviceToHost);
    if (hipGetLastError() != hipSuccess)
	{
		fprintf(stderr, "Cuda error: Memcpy to host failed\n");
		return;
	}

    hipFree(d_vector);
}
void transpose(hipComplex *h_matrix, const unsigned int width, const unsigned int batch)
{
    hipComplex *d_matrix, *d_out, curr;

    hipMalloc((void**)&d_matrix, sizeof(hipComplex)*width*batch);
    if (hipGetLastError() != hipSuccess)
	{
		fprintf(stderr, "Cuda error: Failed to allocate memory for matrix\n");
		return;
	}

    hipMalloc((void**)&d_out, sizeof(hipComplex)*width*batch);
    if (hipGetLastError() != hipSuccess)
	{
		fprintf(stderr, "Cuda error: Failed to allocate memory for output\n");
		return;
	}

    //Copying matrix onto device
    hipMemcpy(d_matrix, h_matrix, sizeof(hipComplex)*width*batch,
               hipMemcpyHostToDevice);
    if (hipGetLastError() != hipSuccess)
	{
		fprintf(stderr, "Cuda error: Memcpy to device failed\n");
		return;
	}

    dim3 threadsPerBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 numOfBlocks(width/(threadsPerBlock.x + BLOCK_SIZE) + 1, batch/threadsPerBlock.y + 1);

    transpose_kernel<<<numOfBlocks, threadsPerBlock>>>(d_matrix, d_out, width, batch);

    hipMemcpy(h_matrix, d_out, sizeof(hipComplex)*width*batch, hipMemcpyDeviceToHost);
    if (hipGetLastError() != hipSuccess)
	{
		fprintf(stderr, "Cuda error: Memcpy to host failed\n");
		return;
	}

    hipFree(d_matrix);
    hipFree(d_out);

    return;
}
void fftshift(hipComplex *h_signal, const unsigned int width, const unsigned int batch, const int dim)
{
    hipComplex *d_signal, *d_out, curr;

    hipMalloc((void**)&d_signal, sizeof(hipComplex)*width*batch);
    if (hipGetLastError() != hipSuccess)
	{
		fprintf(stderr, "Cuda error: Failed to allocate memory for signal\n");
		return;
	}

    hipMalloc((void**)&d_out, sizeof(hipComplex)*width*batch);
    if (hipGetLastError() != hipSuccess)
	{
		fprintf(stderr, "Cuda error: Failed to allocate memory for output\n");
		return;
	}

    //Copying matrix onto device
    hipMemcpy(d_signal, h_signal, sizeof(hipComplex)*width*batch,
               hipMemcpyHostToDevice);

    dim3 threadsPerBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 numOfBlocks(width/threadsPerBlock.x + 1, batch/threadsPerBlock.y + 1);

    fftshift_kernel<<<numOfBlocks, threadsPerBlock>>>(d_signal, d_out, width, batch, dim);

    hipMemcpy(h_signal, d_out, sizeof(hipComplex)*width*batch,
               hipMemcpyDeviceToHost);

    hipFree(d_out);
    hipFree(d_signal);
    
    return;
}
void mapMaker(hipComplex *s_M, hipComplex *mapOut, const unsigned int width, const unsigned int batch, const unsigned int mapLength, const unsigned int mapWidth)
{// Multiplies vector of certain width by each row in matrix
    hipComplex *dS_M, *dMapOut;

    //Allocating memory on GPU
    hipMalloc((void**)&dS_M, sizeof(hipComplex)*width*batch);
	if (hipGetLastError() != hipSuccess)
	{
		fprintf(stderr, "Cuda error: Failed to allocate data\n");
		return;
	}

    hipMalloc((void**)&dMapOut, sizeof(hipComplex)*mapLength*mapWidth);
    if (hipGetLastError() != hipSuccess)
	{
		fprintf(stderr, "Cuda error: Failed to allocate data\n");
		return;
	}
    //Finished Allocation

    //Copying matrix onto device
    hipMemcpy(dS_M, s_M, sizeof(hipComplex)*width*batch, hipMemcpyHostToDevice);

    dim3 threadsPerBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 numOfBlocks(mapLength/threadsPerBlock.x + 1, mapWidth/threadsPerBlock.y + 1);

    map_kernel<<<numOfBlocks, threadsPerBlock>>>(dS_M, dMapOut, width, batch, mapLength, mapWidth);

    hipMemcpy(mapOut, dMapOut, sizeof(hipComplex)*mapLength*mapWidth, hipMemcpyDeviceToHost);

    //Printing map values to console.
    hipFree(dS_M);
    hipFree(dMapOut);
    return;
}
void mat_vec_mult(hipComplex *h_matrix, hipComplex *h_vector, hipComplex *h_out, const unsigned int width, const unsigned int batch)
{
    hipComplex *d_matrix, *d_vector, *d_out;

    hipMalloc((void**)&d_matrix, sizeof(hipComplex)*width*batch);
    if (hipGetLastError() != hipSuccess)
	{
		fprintf(stderr, "Cuda error: Failed to allocate memory for matrix\n");
		return;
	}

    hipMalloc((void**)&d_vector, sizeof(hipComplex)*width);
    if (hipGetLastError() != hipSuccess)
	{
		fprintf(stderr, "Cuda error: Failed to allocate memory for matrix\n");
		return;
	}

    hipMalloc((void**)&d_out, sizeof(hipComplex)*width*batch);
    if (hipGetLastError() != hipSuccess)
	{
		fprintf(stderr, "Cuda error: Failed to allocate memory for matrix\n");
		return;
	}

    hipMemcpy(d_matrix, h_matrix, sizeof(hipComplex)*width*batch,
               hipMemcpyHostToDevice);
    if (hipGetLastError() != hipSuccess)
	{
		fprintf(stderr, "Cuda error: Memcpy to device failed\n");
		return;
	}

    hipMemcpy(d_vector, h_vector, sizeof(hipComplex)*width,
               hipMemcpyHostToDevice);
    if (hipGetLastError() != hipSuccess)
	{
		fprintf(stderr, "Cuda error: Memcpy to device failed\n");
		return;
    }

    dim3 threadsPerBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 numOfBlocks(width/threadsPerBlock.x + 1, batch/threadsPerBlock.y + 1);

    mat_vec_mult_kernel<<<numOfBlocks, threadsPerBlock>>>(d_matrix, d_vector, d_out, width, batch);

    hipMemcpy(h_out, d_out, sizeof(hipComplex)*width*batch, hipMemcpyDeviceToHost);

    hipFree(d_out);
    hipFree(d_matrix);
    hipFree(d_vector);
}

// Produces Compression Constants
void comp_decomp(const float Xc, hipComplex *uc, const int length,  hipComplex *u, const int u_len, hipComplex *k, const int width)
{
    hipComplex *compression, *decompression;

    compression = (hipComplex *)malloc(sizeof(hipComplex)*length*width);
    decompression = (hipComplex *)malloc(sizeof(hipComplex)*u_len*width);

    // fftshift uc
    fftshift(uc, length, 1, 0);
    
    // Square each element
    square(uc, uc, length, 1);
    square(u, u, u_len, 1);

    // add constant to vector
    sca_vec_add(Xc*Xc, uc, length, 1, 1);
    sca_vec_add(Xc*Xc, u, u_len, 1, 1);

    // sqrt(abs complex vector)
    sqrt_abs(uc, uc, length, 1);
    sqrt_abs(u, u, u_len, 1);

    // subtract contant from vector
    sca_vec_add(-1.0*Xc, uc, length, 1, 1.0);
    // Xc - u
    sca_vec_add(Xc, u, u_len, 1, -1.0);

    // change real vector imaginary vector
    real_to_imag(k, k, width, 1);
    sca_vec_mult(2.0, k, width, 1);

    // mult vec vec to matrix
    vec_vec_mat(uc, k, compression, length, width);
    vec_vec_mat(k, u, decompression, width, u_len);

    // exp mat

}

int main()
{
	//new code
	hipDeviceReset();
	hipSetDevice(0);

	// Get all data
	ifstream fastTimeFilter ("fastTimeFilter.csv");
	ifstream imagsRaw ("imagsRaw.csv");
	ifstream realsRaw ("realsRaw.csv");
	// End of data read

	float d, i;
    int x,y;

    //Dimensions of sRaw data
    int width = 438;
    int batch = 160;

    //Dimensions of final map
    int mapLength = 382;
    int mapWidth  = 266;

	hipComplex curr, *sRaw, *d_sRaw, *signal, *d_signal, *sM, *mapOut, *out_signal;
    hipfftHandle plan;
	
	signal = (hipComplex *)malloc(sizeof(hipComplex)*width);
	sRaw   = (hipComplex *)malloc(sizeof(hipComplex)*width*batch);
    out_signal = (hipComplex *)malloc(sizeof(hipComplex)*width*batch);
    
    //Copying Data from CSV files into memory
    string value;
    list<string> values;
    int count = width - 1;

    while ( fastTimeFilter.good() )
    {
        getline ( fastTimeFilter, value, ',' ); // read a string until next comma: http://www.cplusplus.com/reference/string/getline/
        if (value.find('\n') != string::npos) {
            split_line(value, "\n", values);
        } else {
            values.push_back(value);
        }
    }
    //From fast time filter we get p*(-t)
    list<string>::const_iterator it = values.begin();
    for (it = values.begin(); it != values.end(); it++) {
        string tmp = *it;
        d = stof(tmp.c_str(), NULL);
		it++;
		tmp = *it;
		i = stof(tmp.c_str(), NULL);
		signal[count].x = d;
		signal[count].y = -1 * i;
		count--;
        //cout << "Double val: " << right << showpoint << d << endl;
    }
	
	string value1;
    list<string> values1;
    while ( imagsRaw.good() )
    {
        getline ( imagsRaw, value1, ',' ); // read a string until next comma: http://www.cplusplus.com/reference/string/getline/
        if (value1.find('\n') != string::npos) {
            split_line(value1, "\n", values1);
        } else {
            values1.push_back(value1);
        }
    }

    it = values1.begin();
    count = 0;
    for (it = values1.begin(); it != values1.end(); it++) {
        string tmp = *it;
        d = stof(tmp.c_str(), NULL);
        sRaw[count].y = d;
        count++;
        //cout << "Double val: " << right << showpoint << d << endl;
    }

    string value2;
    list<string> values2;
    while ( realsRaw.good() )
    {
        getline ( realsRaw, value2, ',' ); // read a string until next comma: http://www.cplusplus.com/reference/string/getline/
        if (value2.find('\n') != string::npos) {
            split_line(value2, "\n", values2);
        } else {
            values2.push_back(value2);
        }
    }

    it = values2.begin();
    count = 0;
    for (it = values2.begin(); it != values2.end(); it++) {
        string tmp = *it;
        d = stof(tmp.c_str(), NULL);
		sRaw[count].x = d;
		count++;
        //cout << "Double val: " << right << showpoint << d << endl;
    }
	//Done Reading in values from files.

    //---------------------------------------------------------------------------------------------------------------------

    // Output for convolution
    //sM = (hipComplex *)malloc(sizeof(hipComplex)*tots*batch);

	//convolveWithCuda(sRaw, signal, sM, width, batch);

    transpose(sRaw, width, batch);

    hipMalloc((void**)&d_sRaw, sizeof(hipComplex)*width*batch);
    if (hipGetLastError() != hipSuccess)
	{
		fprintf(stderr, "Cuda error: Failed to allocate data\n");
		return;
	}

    hipMemcpy(d_sRaw, sRaw, sizeof(hipComplex)*width*batch,
               hipMemcpyHostToDevice);
    if (hipGetLastError() != hipSuccess)
	{
		fprintf(stderr, "Cuda error: Memcpy to device failed\n");
		return;
	}

    hipMalloc((void**)&d_signal, sizeof(hipComplex)*width);
    if (hipGetLastError() != hipSuccess)
	{
		fprintf(stderr, "Cuda error: Failed to allocate data\n");
		return;
	}

    hipMemcpy(d_signal, signal, sizeof(hipComplex)*width,
               hipMemcpyHostToDevice);
    if (hipGetLastError() != hipSuccess)
	{
		fprintf(stderr, "Cuda error: Memcpy to device for signal\n");
		return;
	}

    hipfftPlan1d(&plan, width, HIPFFT_C2C, batch);
    hipfftExecC2C(plan, d_sRaw, d_sRaw, HIPFFT_FORWARD);
    if (hipGetLastError() != hipSuccess)
	{
		fprintf(stderr, "Cuda error: cufft failed\n");
		return;
	}

    hipMemcpy(sRaw, d_sRaw, sizeof(hipComplex)*width*batch,
               hipMemcpyDeviceToHost);
    if (hipGetLastError() != hipSuccess)
	{
		fprintf(stderr, "Cuda error: Memcpy from device failed\n");
		return;
	}
    hipfftDestroy(plan);
    
    fftshift(sRaw, width, batch, 2);

    fftshift(signal, width, 1, 0);

    mat_vec_mult(sRaw, signal, sRaw, width, batch);

    for(int x = 0; x < batch; x++)
    {
        for(int y = 0; y < width; y++)
        {
            curr = sRaw[x* width + y];
            printf("%g + (%gi), ", hipCrealf(curr), hipCimagf(curr));
        }
        cout << endl;
    }

    hipFree(d_sRaw);
    hipFree(d_signal);
    free(sRaw);
    free(signal);
    free(out_signal);
	return 0;
}
