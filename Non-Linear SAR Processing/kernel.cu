#include "hip/hip_runtime.h"
// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <iostream>
#include <fstream>
#include <cstdlib>
#include <list>

// CUDA includes
#include <hip/hip_runtime.h>
#include ""

// Utilities and timing functions
#include <helper_functions.h>    // includes hip/hip_runtime.h and hip/hip_runtime_api.h

// CUDA helper functions
#include <hip/hip_runtime_api.h>         // helper functions for CUDA error check

// CUDA Libraries
#include <hipfft/hipfft.h>
#include <hipfft/hipfftw.h>
#include <hiprand/hiprand_kernel.h>
#include <math_functions.h>

#include <hip/hip_vector_types.h>
#include "utils.h"

#include <array>

#include "builtin_types.h"
#include "host_defines.h"

using namespace std;

#define BLOCK_SIZE 32

#define PI 3.14159265358979323846

void split_line(string& line, string delim, list<string>& values)
{
    size_t pos = 0;
    while ((pos = line.find(delim, (pos + 1))) != string::npos) {
        string p = line.substr(0, pos);
        values.push_back(p);
        line = line.substr(pos + 1);
    }

    if (!line.empty()) {
        values.push_back(line);
    }
}

void csv_real_reader(string filename, hipComplex *signal, bool isReal, bool zero)
{
    float d;
    int count = 0;
    ifstream csv (filename);
    
    //Copying Data from CSV files into memory
    string value;
    list<string> values;
    while ( csv.good() )
    {
        // read a string until next comma: http://www.cplusplus.com/reference/string/getline/
        getline ( csv, value, ',' ); 
        if (value.find('\n') != string::npos)
            split_line(value, "\n", values);
        else
            values.push_back(value);
    }

    list<string>::const_iterator it = values.begin();
    for (it = values.begin(); it != values.end(); it++) {
        string tmp = *it;
        d = stof(tmp.c_str(), NULL);
        if (isReal) {
		    signal[count].x = d;
            if (zero) {
                signal[count].y = 0;
            }
        } else {
            signal[count].y = d;
            if (zero) {
                signal[count].x = 0;
            }
        }
		count++;
    }
}

static __inline__ __device__ float atomicfAdd(float *address, float val)
{
  return __fAtomicAdd(address, val);
}

//kernel functions
__global__ void square_kernel(hipComplex *d_vector, hipComplex *d_out, const int length, const int width)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row >= length || col >= width) {return;}

    d_out[width*row + col].x = hipCabsf(d_vector[width*row + col]) * hipCabsf(d_vector[width*row + col]);
    d_out[width*row + col].y = 0.0;
}
__global__ void sqrt_abs_kernel(hipComplex *d_in, hipComplex *d_out, const int length, const int width)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row >= length || col >= width) {return;}

    d_out[width*row + col].x = sqrtf(hipCabsf(d_in[width*row + col]));
    d_out[width*row + col].y = 0;
}
__global__ void exp_mat_kernel(hipComplex *d_in, hipComplex *d_out, const int length, const int width)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row >= length || col >= width) {return;}

    float s, c;
    float e = expf(d_in[width*row + col].x);
    sincosf(d_in[width*row + col].y, &s, &c);

    d_out[width*row + col].x = c * e;
    d_out[width*row + col].y = s * e;
}
__global__ void real_to_imag_kernel(hipComplex *d_in, hipComplex *d_out, const int length, const int width)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row >= length || col >= width) {return;}
    
    d_out[width*row + col].x = 0;
    d_out[width*row + col].y = d_in[width*row + col].x;
}
__global__ void vec_vec_mult_kernel(hipComplex *d_vec1, hipComplex *d_vec2, hipComplex *d_out, const int length, const int width)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    int newRow, ind;

    if (row*BLOCK_SIZE >= length || col >= width) {return;}

    for (int i = 0; i < BLOCK_SIZE; i++)
    {
        newRow = (BLOCK_SIZE*row) + i;
        if(newRow < length)
        {
            ind = col + width*newRow;
            d_out[ind] = hipCmulf(d_vec1[ind], d_vec2[ind]);
        }
    }
}
__global__ void vec_vec_mat_kernel(hipComplex *d_vec1, hipComplex *d_vec2, hipComplex *d_out, const int len_1, const int len_2)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row >= len_1 || col >= len_2) {return;}

    d_out[len_2*row + col] = hipCmulf(d_vec1[row], d_vec2[col]);
}
__global__ void sca_vec_add_kernel(const double K, hipComplex *d_vector, const int length, const int width, const double M)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row >= length || col >= width) {return;}

    d_vector[width*row + col].x = K + M*d_vector[width*row + col].x;
    d_vector[width*row + col].y = M*d_vector[width*row + col].y;
}
__global__ void sca_vec_mult_kernel(const double K, hipComplex *d_vector, const int length, const int width)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    int newRow;

    if (row*BLOCK_SIZE >= length || col >= width) {return;}

    for (int i = 0; i < BLOCK_SIZE; i++)
    {
        newRow = (BLOCK_SIZE*row) + i;
        if(newRow < length)
        {
            d_vector[col + width*newRow].x *= K;
            d_vector[col + width*newRow].y *= K;
        }
    }
}
__global__ void transpose_kernel(hipComplex *d_matrix, hipComplex *d_out, const int length, const int width)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row >= width || col >= length) {return;}

    d_out[width*col + row] = d_matrix[length*row + col];
        
    return;
}
__global__ void fftshift_kernel(hipComplex *d_signal, hipComplex *d_out, const int width, const int batch, const int dim)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    
    if ((row >= batch) || (col >= width))
        return;

    int newcol = col;
    int newrow = row;

    if (dim != 1) 
        newcol = (col + width/2 +3) % width;

    if (dim != 2) 
        newrow = (row + batch/2 +3) % batch;
    
    d_out[newcol + newrow*width] = d_signal[col + row*width];
}
__global__ void map_kernel(hipComplex *s_M, hipComplex *out, const int width, const int batch, const int max_x, const int max_y)
{   /*s_M is the fast-time matched filtered SAR signal
     *Because this signal is in discrete time and uses indices 
     *instead of values, we have to modify the time delay
     *function to fit between the range 1 and the width of s_M
    */

    // We will run a total of max_x * max_y threads,
    // looping through all possible slow time locations
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if(max_y - 1 < row || max_x - 1 < col) return;

    int u;
    int delay;
    hipComplex val;
    val.x= 0;
    val.y =0;
    int uNormal = max_y/batch;
    for(u = 0; u < batch; u++)
    {   // The number 1.8821 is width/max(s_m), this will be changed later
        // It is used to evenly distribute the magnitudes of the map
        delay = lround(hypot( (double) row, (double) (col - u*uNormal))*0.01281);
        val.x += s_M[u*width + delay].x;
        val.y += s_M[u*width + delay].y;        
    }
    out[row*max_x + col] = val;
    return;
}
__global__ void mat_vec_mult_kernel(hipComplex *matrix, hipComplex *vector, hipComplex *out, const int width, const int batch)
{
	int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row >= batch || col >= width) {return;}

    out[row*width + col] = hipCmulf(matrix[row*width + col], vector[col]);
}
__global__ void pad_kernel(hipComplex *d_in, hipComplex *d_out, const int length, const int width, const int padInd, const int padLength)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (row >= width || col >= (padLength+length)) {return;}
    int newind = row*(length+padLength) + col;

    if (col < padInd)
        d_out[newind] = d_in[row*length + col];
    else if (col < padLength + padInd)
        d_out[newind] = make_hipComplex(0,0);
    else 
        d_out[newind] = d_in[row*length + col - padLength];

    return;
}
__global__ void vec_copy2mat_kernel(hipComplex *d_in, hipComplex *d_out, const int length, const int width)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row >= width || col >= length) {return;}

    d_out[length*row + col] = d_in[col];

    return;
}
__global__ void vec_vec_add_kernel(hipComplex *d_mat1, hipComplex *d_mat2, hipComplex *d_out, const int length, const int width)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row >= width || col >= length) {return;}

    int ind = length*row + col;

    d_out[ind] = hipCaddf(d_mat1[ind], d_mat2[ind]);

    return;
}
__global__ void sca_max_kernel(float K, hipComplex *d_in, hipComplex *d_out, const int length, const int width)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row >= width || col >= length) {return;}

    int ind = length*row + col;

    if (d_in[ind].x > K) 
    { 
        d_out[ind] = d_in[ind];
    }
    else 
    {
        d_out[ind].x = K;
        d_out[ind].y = 0.0;
    }

    return;
}
__global__ void is_pos_kernel(hipComplex *d_in, hipComplex *d_out, const int length, const int width)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row >= width || col >= length) {return;}

    int ind = length*row + col;

    if(d_in[ind].x > 0.0) 
        d_out[ind] = make_hipComplex(1.0, 0.0);
    else
        d_out[ind] = make_hipComplex(0.0, 0.0);
}
__global__ void round_vec_kernel(hipComplex *d_in, hipComplex *d_out, const int length, const int width)
{
    //d_in is real
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;
     
    if (row >= width || col >= length) {return;}

    int ind = length*row + col;

    d_out[ind] = make_hipComplex(roundf(hipCabsf(d_in[ind])), 0.0);

    return;
}
__global__ void cuComplex2Int_kernel(hipComplex *h_in, int *h_out, const int length, const int width)
{
    int col = blockDim.x * blockIdx.x + threadIdx.x;
    int row = blockDim.y * blockIdx.y + threadIdx.y;

     if (row >= width || col >= length) {return;}

     int ind = length*row + col;

     h_out[ind] = lroundf(hipCabsf(h_in[ind]));

    return;
}
__global__ void cuComplex2float_kernel(hipComplex *h_in, float *h_out, const int length, const int width)
{
    int col = blockDim.x * blockIdx.x + threadIdx.x;
    int row = blockDim.y * blockIdx.y + threadIdx.y;

     if (row >= width || col >= length) {return;}

     int ind = length*row + col;

     h_out[ind] = hipCabsf(h_in[ind]);

    return;
}
__global__ void float2cuComplex_kernel(float *h_in, hipComplex *h_out, const int length, const int width)
{
    int col = blockDim.x * blockIdx.x + threadIdx.x;
    int row = blockDim.y * blockIdx.y + threadIdx.y;

     if (row >= width || col >= length) {return;}

     int ind = length*row + col;

     h_out[ind].x = h_in[ind];
     h_out[ind].y = 0.0;

    return;
}
__global__ void make_gridValues_kernel(hipComplex *gridValues, const double kxMin, const int mapWidth, const int nInterpSidelobes, const double dkx)
{
     int col = blockDim.x * blockIdx.x + threadIdx.x;

     if (col >= mapWidth) {return;}

     gridValues[col].x = kxMin + dkx*(col-nInterpSidelobes-3);
     gridValues[col].y = 0.0;

     return;
}
__global__ void zero_mat_kernel(hipComplex *d_mat, const int length, const int width)
{
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;
 
    if (col >= length || row >= width) {return;}

    int ind = length*row + col;

    d_mat[ind].x = 0.0;
    d_mat[ind].y = 0.0;

    return;
}
__global__ void spatial_inter_kernel(hipComplex *filteredSignal, float *kx, float *GridValues, hipComplex *rowidx, const int length, const int mapLength, const int mapWidth, const int nInterpSideLobes, const float dkx, const float kxs, hipComplex *out, hipComplex *slicerange)
{
    unsigned int col = blockDim.x * blockIdx.x + threadIdx.x;
    unsigned int row = blockDim.y * blockIdx.y + threadIdx.y;
    unsigned int stack = blockDim.z * blockIdx.z + threadIdx.z;
    
    if ( col >= length || row >= mapWidth || stack >= (2*nInterpSideLobes - 1)) {return;}
    //()
    float ham;

    long unsigned int ind = length*row + col;
    int idxout = lroundf(rowidx[ind].x) + stack;
    
    slicerange[ind].x = GridValues[idxout] - kx[ind];
    slicerange[ind].y = 0.0;

    if (slicerange[ind].x/dkx == 0.0f)
        ham = 0.54 + 0.46*cos( slicerange[ind].x * PI/kxs );
    else
        ham = (0.54 + 0.46*cos( slicerange[ind].x * PI/kxs ))*sin(PI * slicerange[ind].x/dkx)/(PI * slicerange[ind].x/dkx);

    long unsigned int image_ind = 266*row + idxout;

    atomicfAdd((float *)&(out[image_ind].x), filteredSignal[ind].x*ham);
    atomicfAdd((float *)&(out[image_ind].y), filteredSignal[ind].y*ham);

    return;
}

// kernel helpers
void square(hipComplex *h_vector, hipComplex *h_out, const int length, const int width)
{
    hipComplex *d_vector, *d_out;

    checkCudaErrors(hipMalloc((void**)&d_vector, sizeof(hipComplex)*width*length));

    checkCudaErrors(hipMalloc((void**)&d_out, sizeof(hipComplex)*width*length));
    
    checkCudaErrors(hipMemcpy(d_vector, h_vector, sizeof(hipComplex)*width*length,
               hipMemcpyHostToDevice));

    dim3 threadsPerBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 numOfBlocks(width/threadsPerBlock.x + 1, length/threadsPerBlock.y + 1);

    square_kernel<<<numOfBlocks, threadsPerBlock>>>(d_vector, d_out, length, width);

    checkCudaErrors(hipMemcpy(h_out, d_out, sizeof(hipComplex)*width*length,
               hipMemcpyDeviceToHost));
    
    checkCudaErrors(hipFree(d_out));
    checkCudaErrors(hipFree(d_vector));
}
void sqrt_abs(hipComplex *h_in, hipComplex *h_out, const int length, const int width)
{
    hipComplex *d_in, *d_out;

    checkCudaErrors(hipMalloc((void**)&d_in, sizeof(hipComplex)*width*length));

    checkCudaErrors(hipMalloc((void**)&d_out, sizeof(hipComplex)*width*length));

    checkCudaErrors(hipMemcpy(d_in, h_in, sizeof(hipComplex)*width*length, hipMemcpyHostToDevice));

    dim3 threadsPerBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 numOfBlocks(width/threadsPerBlock.x + 1, length/threadsPerBlock.y + 1);

    sqrt_abs_kernel<<<numOfBlocks, threadsPerBlock>>>(d_in, d_out, length, width);

    checkCudaErrors(hipMemcpy(h_out, d_out, sizeof(hipComplex)*width*length, hipMemcpyDeviceToHost));

    checkCudaErrors(hipFree(d_in));
    checkCudaErrors(hipFree(d_out));
}
void exp_mat(hipComplex *h_in, hipComplex *h_out, const int length, const int width)
{
    hipComplex *d_in, *d_out;

    checkCudaErrors(hipMalloc((void**)&d_in, sizeof(hipComplex)*width*length));
    
    checkCudaErrors(hipMalloc((void**)&d_out, sizeof(hipComplex)*width*length));
    
    checkCudaErrors(hipMemcpy(d_in, h_in, sizeof(hipComplex)*width*length,
               hipMemcpyHostToDevice));
    

    dim3 threadsPerBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 numOfBlocks(width/threadsPerBlock.x + 1, length/threadsPerBlock.y + 1);

    exp_mat_kernel<<<numOfBlocks, threadsPerBlock>>>(d_in, d_out, length, width);

    checkCudaErrors(hipMemcpy(h_out, d_out, sizeof(hipComplex)*width*length, hipMemcpyDeviceToHost));

    checkCudaErrors(hipFree(d_in));
    checkCudaErrors(hipFree(d_out));
}
void real_to_imag(hipComplex *h_in, hipComplex *h_out, const int length, const int width)
{
    hipComplex *d_in, *d_out;

    checkCudaErrors(hipMalloc((void**)&d_in, sizeof(hipComplex)*width*length));

    checkCudaErrors(hipMalloc((void**)&d_out, sizeof(hipComplex)*width*length));
    
    checkCudaErrors(hipMemcpy(d_in, h_in, sizeof(hipComplex)*width*length, hipMemcpyHostToDevice));
    
    dim3 threadsPerBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 numOfBlocks(width/threadsPerBlock.x + 1, length/threadsPerBlock.y + 1);

    real_to_imag_kernel<<<numOfBlocks, threadsPerBlock>>>(d_in, d_out, length, width);

    checkCudaErrors(hipMemcpy(h_out, d_out, sizeof(hipComplex)*width*length, hipMemcpyDeviceToHost));
    
    checkCudaErrors(hipFree(d_in));
    checkCudaErrors(hipFree(d_out));
}
void vec_vec_mult(hipComplex *h_vec1, hipComplex *h_vec2, hipComplex *h_out, const int length, const int width)
{
    //Element wise multiplication of 2 vectors, output is placed in h_vec1
    hipComplex *d_vec1, *d_vec2, *d_out;

    checkCudaErrors(hipMalloc((void**)&d_vec1, sizeof(hipComplex)*width*length));
    
    checkCudaErrors(hipMalloc((void**)&d_vec2, sizeof(hipComplex)*width*length));

    checkCudaErrors(hipMalloc((void**)&d_out, sizeof(hipComplex)*width*length));

    //Copying vectors onto device
    checkCudaErrors(hipMemcpy(d_vec1, h_vec1, sizeof(hipComplex)*width*length, hipMemcpyHostToDevice));
    
    checkCudaErrors(hipMemcpy(d_vec2, h_vec2, sizeof(hipComplex)*width*length, hipMemcpyHostToDevice));
    
    dim3 threadsPerBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 numOfBlocks(width/threadsPerBlock.x + 1, length/(threadsPerBlock.y + BLOCK_SIZE) + 1);

    vec_vec_mult_kernel<<<numOfBlocks, threadsPerBlock>>>(d_vec1, d_vec2, d_out, length, width);

    checkCudaErrors(hipMemcpy(h_out, d_out, sizeof(hipComplex)*width*length, hipMemcpyDeviceToHost));
    
    checkCudaErrors(hipFree(d_vec1));
    checkCudaErrors(hipFree(d_vec2));
    checkCudaErrors(hipFree(d_out));
    return;
}
void vec_vec_mat(hipComplex *h_vec1, hipComplex *h_vec2, hipComplex *h_out, const int len_1, const int len_2)
{
    hipComplex *d_vec1, *d_vec2, *d_out;

    checkCudaErrors(hipMalloc((void**)&d_vec1, sizeof(hipComplex)*len_1));

    checkCudaErrors(hipMalloc((void**)&d_vec2, sizeof(hipComplex)*len_2));
    
    checkCudaErrors(hipMalloc((void**)&d_out, sizeof(hipComplex)*len_1*len_2));
    
    //Copying vectors onto device
    checkCudaErrors(hipMemcpy(d_vec1, h_vec1, sizeof(hipComplex)*len_1, hipMemcpyHostToDevice));
    
    checkCudaErrors(hipMemcpy(d_vec2, h_vec2, sizeof(hipComplex)*len_2, hipMemcpyHostToDevice));
    
    dim3 threadsPerBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 numOfBlocks(len_2/threadsPerBlock.x + 1, len_1/threadsPerBlock.y + 1);

    vec_vec_mat_kernel<<<numOfBlocks, threadsPerBlock>>>(d_vec1, d_vec2, d_out, len_1, len_2);

    checkCudaErrors(hipMemcpy(h_out, d_out, sizeof(hipComplex)*len_2*len_1, hipMemcpyDeviceToHost));
    
    checkCudaErrors(hipFree(d_out));
    checkCudaErrors(hipFree(d_vec1));
    checkCudaErrors(hipFree(d_vec2));
}
void sca_vec_add(const double K, hipComplex *h_vector, const int length, const int width, const double M)
{
    hipComplex *d_vector;

    hipMalloc((void**)&d_vector, sizeof(hipComplex)*width*length);
    if (hipGetLastError() != hipSuccess)
	{
		fprintf(stderr, "Cuda error: Failed to allocate memory for matrix\n");
		return;
	}

    checkCudaErrors(hipMemcpy(d_vector, h_vector, sizeof(hipComplex)*width*length,
               hipMemcpyHostToDevice));
    if (hipGetLastError() != hipSuccess)
	{
		fprintf(stderr, "Cuda error: Memcpy to device failed\n");
		return;
	}

    dim3 threadsPerBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 numOfBlocks(width/threadsPerBlock.x + 1, length/threadsPerBlock.y + 1);

    sca_vec_add_kernel<<<numOfBlocks, threadsPerBlock>>>(K, d_vector, length, width, M);

    checkCudaErrors(hipMemcpy(h_vector, d_vector, sizeof(hipComplex)*width*length,
               hipMemcpyDeviceToHost));
    if (hipGetLastError() != hipSuccess)
	{
		fprintf(stderr, "Cuda error: Memcpy to device failed\n");
		return;
	}

    hipFree(d_vector);
}
void sca_vec_mult(const double K, hipComplex *h_vector, const int length, const int width)
{
    hipComplex *d_vector;

    hipMalloc((void**)&d_vector, sizeof(hipComplex)*width*length);
    if (hipGetLastError() != hipSuccess)
	{
		fprintf(stderr, "Cuda error: Failed to allocate memory for matrix\n");
		return;
	}

    //Copying vector onto device
    checkCudaErrors(hipMemcpy(d_vector, h_vector, sizeof(hipComplex)*width*length,
               hipMemcpyHostToDevice));
   
    dim3 threadsPerBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 numOfBlocks(width/threadsPerBlock.x + 1, length/(threadsPerBlock.y + BLOCK_SIZE) + 1);

    sca_vec_mult_kernel<<<numOfBlocks, threadsPerBlock>>>(K, d_vector, length, width);

    checkCudaErrors(hipMemcpy(h_vector, d_vector, sizeof(hipComplex)*width*length, hipMemcpyDeviceToHost));
    
    hipFree(d_vector);
}
void transpose(hipComplex *h_matrix, const int width, const int batch)
{
    hipComplex *d_matrix, *d_out;

    hipMalloc((void**)&d_matrix, sizeof(hipComplex)*width*batch);
    if (hipGetLastError() != hipSuccess)
	{
		fprintf(stderr, "Cuda error: Failed to allocate memory for matrix\n");
		return;
	}

    hipMalloc((void**)&d_out, sizeof(hipComplex)*width*batch);
    if (hipGetLastError() != hipSuccess)
	{
		fprintf(stderr, "Cuda error: Failed to allocate memory for output\n");
		return;
	}

    //Copying matrix onto device
    checkCudaErrors(hipMemcpy(d_matrix, h_matrix, sizeof(hipComplex)*width*batch,
               hipMemcpyHostToDevice));
    if (hipGetLastError() != hipSuccess)
	{
		fprintf(stderr, "Cuda error: Memcpy to device failed\n");
		return;
	}

    dim3 threadsPerBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 numOfBlocks(width/threadsPerBlock.x + 1, batch/threadsPerBlock.y + 1);

    transpose_kernel<<<numOfBlocks, threadsPerBlock>>>(d_matrix, d_out, width, batch);

    checkCudaErrors(hipMemcpy(h_matrix, d_out, sizeof(hipComplex)*width*batch, hipMemcpyDeviceToHost));
    
    checkCudaErrors(hipFree(d_matrix));
    checkCudaErrors(hipFree(d_out));

    return;
}
void fftshift(hipComplex *h_signal, hipComplex *h_out, const int width, const int batch, const int dim)
{
    hipComplex *d_signal, *d_out;

    hipMalloc((void**)&d_signal, sizeof(hipComplex)*width*batch);
    if (hipGetLastError() != hipSuccess)
	{
		fprintf(stderr, "Cuda error: Failed to allocate memory for signal\n");
		return;
	}

    hipMalloc((void**)&d_out, sizeof(hipComplex)*width*batch);
    if (hipGetLastError() != hipSuccess)
	{
		fprintf(stderr, "Cuda error: Failed to allocate memory for output\n");
		return;
	}

    //Copying matrix onto device
    checkCudaErrors(hipMemcpy(d_signal, h_signal, sizeof(hipComplex)*width*batch,
               hipMemcpyHostToDevice));

    dim3 threadsPerBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 numOfBlocks(width/threadsPerBlock.x + 1, batch/threadsPerBlock.y + 1);

    fftshift_kernel<<<numOfBlocks, threadsPerBlock>>>(d_signal, d_out, width, batch, dim);

    checkCudaErrors(hipMemcpy(h_out, d_out, sizeof(hipComplex)*width*batch,
               hipMemcpyDeviceToHost));

    hipFree(d_out);
    hipFree(d_signal);
    
    return;
}
void mapMaker(hipComplex *s_M, hipComplex *mapOut, const int width, const int batch, const int mapLength, const int mapWidth)
{// Multiplies vector of certain width by each row in matrix
    hipComplex *dS_M, *dMapOut;

    //Allocating memory on GPU
    checkCudaErrors(hipMalloc((void**)&dS_M, sizeof(hipComplex)*width*batch));

    checkCudaErrors(hipMalloc((void**)&dMapOut, sizeof(hipComplex)*mapLength*mapWidth));
    //Finished Allocation

    //Copying matrix onto device
    checkCudaErrors(hipMemcpy(dS_M, s_M, sizeof(hipComplex)*width*batch, hipMemcpyHostToDevice));

    dim3 threadsPerBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 numOfBlocks(mapLength/threadsPerBlock.x + 1, mapWidth/threadsPerBlock.y + 1);

    map_kernel<<<numOfBlocks, threadsPerBlock>>>(dS_M, dMapOut, width, batch, mapLength, mapWidth);

    checkCudaErrors(hipMemcpy(mapOut, dMapOut, sizeof(hipComplex)*mapLength*mapWidth, hipMemcpyDeviceToHost));

    //Printing map values to console.
    checkCudaErrors(hipFree(dS_M));
    checkCudaErrors(hipFree(dMapOut));
    return;
}
void mat_vec_mult(hipComplex *h_matrix, hipComplex *h_vector, hipComplex *h_out, const int width, const int batch)
{
    hipComplex *d_matrix, *d_vector, *d_out;

    checkCudaErrors(hipMalloc((void**)&d_matrix, sizeof(hipComplex)*width*batch));
    checkCudaErrors(hipMalloc((void**)&d_vector, sizeof(hipComplex)*width));
    checkCudaErrors(hipMalloc((void**)&d_out, sizeof(hipComplex)*width*batch));

    checkCudaErrors(hipMemcpy(d_matrix, h_matrix, sizeof(hipComplex)*width*batch, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_vector, h_vector, sizeof(hipComplex)*width, hipMemcpyHostToDevice));
    
    dim3 threadsPerBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 numOfBlocks(width/threadsPerBlock.x + 1, batch/threadsPerBlock.y + 1);

    mat_vec_mult_kernel<<<numOfBlocks, threadsPerBlock>>>(d_matrix, d_vector, d_out, width, batch);

    checkCudaErrors(hipMemcpy(h_out, d_out, sizeof(hipComplex)*width*batch, hipMemcpyDeviceToHost));

    checkCudaErrors(hipFree(d_out));
    checkCudaErrors(hipFree(d_matrix));
    checkCudaErrors(hipFree(d_vector));
}
void pad(hipComplex *h_in, hipComplex *h_out, const int length, const int width, const int padInd, const int padLength)
{//(sRaw, padded_data, batch, width, batch/2, mapLength - batch);
    
    hipComplex *d_in, *d_out;
    int newLength = padLength+length;

    checkCudaErrors(hipMalloc((void**)&d_in, sizeof(hipComplex)*width*length));
    checkCudaErrors(hipMalloc((void**)&d_out, sizeof(hipComplex)*width*newLength));

    checkCudaErrors(hipMemcpy(d_in, h_in, sizeof(hipComplex)*width*length, hipMemcpyHostToDevice));

    dim3 threadsPerBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 numOfBlocks(newLength/threadsPerBlock.x + 1, width/threadsPerBlock.y + 1);

    pad_kernel<<<numOfBlocks, threadsPerBlock>>>(d_in, d_out, length, width, padInd, padLength);

    checkCudaErrors(hipMemcpy(h_out, d_out, sizeof(hipComplex)*width*newLength, hipMemcpyDeviceToHost));

    checkCudaErrors(hipFree(d_out));
    checkCudaErrors(hipFree(d_in));
    return;
}
void vec_copy2mat(hipComplex *h_in, hipComplex *h_out, const int length, const int width)
{
    //Copies input vector of certain length, over and over to fit in a length X width matrix
    hipComplex *d_in, *d_out;
    
    checkCudaErrors(hipMalloc((void**)&d_in, sizeof(hipComplex)*length));
    checkCudaErrors(hipMalloc((void**)&d_out, sizeof(hipComplex)*width*length));
    
    checkCudaErrors(hipMemcpy(d_in, h_in, sizeof(hipComplex)*length, hipMemcpyHostToDevice));

    dim3 threadsPerBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 numOfBlocks(length/threadsPerBlock.x + 1, width/threadsPerBlock.y + 1);

    vec_copy2mat_kernel<<<numOfBlocks, threadsPerBlock>>>(d_in, d_out, length, width);

    checkCudaErrors(hipMemcpy(h_out, d_out, sizeof(hipComplex)*width*length, hipMemcpyDeviceToHost));

    checkCudaErrors(hipFree(d_out));
    checkCudaErrors(hipFree(d_in));
    return;
}
void vec_vec_add(hipComplex *h_mat1, hipComplex *h_mat2, hipComplex *h_out, const int length, const int width)
{
    hipComplex *d_mat1, *d_mat2, *d_out;

    checkCudaErrors(hipMalloc((void**)&d_mat1, sizeof(hipComplex)*length*width));
    checkCudaErrors(hipMalloc((void**)&d_mat2, sizeof(hipComplex)*length*width));
    checkCudaErrors(hipMalloc((void**)&d_out, sizeof(hipComplex)*width*length));
    
    checkCudaErrors(hipMemcpy(d_mat1, h_mat1, sizeof(hipComplex)*length*width, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_mat2, h_mat2, sizeof(hipComplex)*length*width, hipMemcpyHostToDevice));
    
    dim3 threadsPerBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 numOfBlocks(length/threadsPerBlock.x + 1, width/threadsPerBlock.y + 1);
    
    vec_vec_add_kernel<<<numOfBlocks, threadsPerBlock>>>(d_mat1, d_mat2, d_out, length, width);

    checkCudaErrors(hipMemcpy(h_out, d_out, sizeof(hipComplex)*width*length, hipMemcpyDeviceToHost));
    
    return;
}
void sca_max(float K, hipComplex *h_in, hipComplex *h_out, const int length, const int width)
{
    hipComplex *d_in, *d_out;

    checkCudaErrors(hipMalloc((void**)&d_in, sizeof(hipComplex)*length*width));
    checkCudaErrors(hipMalloc((void**)&d_out, sizeof(hipComplex)*length*width));
    
    checkCudaErrors(hipMemcpy(d_in, h_in, sizeof(hipComplex)*length*width, hipMemcpyHostToDevice));
    
    dim3 threadsPerBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 numOfBlocks(length/threadsPerBlock.x + 1, width/threadsPerBlock.y + 1);
    
    sca_max_kernel<<<numOfBlocks, threadsPerBlock>>>(K, d_in, d_out, length, width);

    checkCudaErrors(hipMemcpy(h_out, d_out, sizeof(hipComplex)*width*length, hipMemcpyDeviceToHost));
    
    checkCudaErrors(hipFree(d_in));
    checkCudaErrors(hipFree(d_out));
    return;
}
void is_pos(hipComplex *h_in, hipComplex *h_out, const int length, const int width)
{
    hipComplex *d_in, *d_out;

    checkCudaErrors(hipMalloc((void**)&d_in, sizeof(hipComplex)*length*width));
    checkCudaErrors(hipMalloc((void**)&d_out, sizeof(hipComplex)*width*length));
    
    checkCudaErrors(hipMemcpy(d_in, h_in, sizeof(hipComplex)*length*width, hipMemcpyHostToDevice));
    
    dim3 threadsPerBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 numOfBlocks(length/threadsPerBlock.x + 1, width/threadsPerBlock.y + 1);

    is_pos_kernel<<<numOfBlocks, threadsPerBlock>>>(d_in, d_out, length, width);

    checkCudaErrors(hipMemcpy(h_out, d_out, sizeof(hipComplex)*width*length, hipMemcpyDeviceToHost));

    checkCudaErrors(hipFree(d_out));
    checkCudaErrors(hipFree(d_in));
    return;
}
void round_vec(hipComplex *h_in, hipComplex *h_out, const int length, const int width)
{
    hipComplex *d_in, *d_out;

    checkCudaErrors(hipMalloc((void**)&d_in, sizeof(hipComplex)*length*width));
    
    checkCudaErrors(hipMalloc((void**)&d_out, sizeof(hipComplex)*width*length));
    
    checkCudaErrors(hipMemcpy(d_in, h_in, sizeof(hipComplex)*length*width, hipMemcpyHostToDevice));

    dim3 threadsPerBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 numOfBlocks(length/threadsPerBlock.x + 1, width/threadsPerBlock.y + 1);

    round_vec_kernel<<<numOfBlocks, threadsPerBlock>>>(d_in, d_out, length, width);
    hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());

    checkCudaErrors(hipMemcpy(h_out, d_out, sizeof(hipComplex)*width*length, hipMemcpyDeviceToHost));
    
    checkCudaErrors(hipFree(d_out));
    checkCudaErrors(hipFree(d_in));
    return;
}
void cuComplex2Int(hipComplex *h_in, int *h_out, const int length, const int width)
{
    hipComplex *d_in;
    int *d_out;

    checkCudaErrors(hipMalloc((void**)&d_in, sizeof(hipComplex)*length*width));
    
    checkCudaErrors(hipMalloc((void**)&d_out, sizeof(int)*length*width));
    
    checkCudaErrors(hipMemcpy(d_in, h_in, sizeof(hipComplex)*length*width, hipMemcpyHostToDevice));

    dim3 threadsPerBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 numOfBlocks(length/threadsPerBlock.x + 1, width/threadsPerBlock.y + 1);

    cuComplex2Int_kernel<<<numOfBlocks, threadsPerBlock>>>(d_in, d_out, length, width);
    hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());

    checkCudaErrors(hipMemcpy(h_out, d_out, sizeof(int)*length*width, hipMemcpyDeviceToHost));
    
    checkCudaErrors(hipFree(d_out));
    checkCudaErrors(hipFree(d_in));
    return;
}
void cuComplex2float(hipComplex *h_in, float *h_out, const int length, const int width)
{
    hipComplex *d_in;
    float *d_out;

    checkCudaErrors(hipMalloc((void**)&d_in, sizeof(hipComplex)*length*width));
    
    checkCudaErrors(hipMalloc((void**)&d_out, sizeof(float)*length*width));
    
    checkCudaErrors(hipMemcpy(d_in, h_in, sizeof(hipComplex)*length*width, hipMemcpyHostToDevice));

    dim3 threadsPerBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 numOfBlocks(length/threadsPerBlock.x + 1, width/threadsPerBlock.y + 1);

    cuComplex2float_kernel<<<numOfBlocks, threadsPerBlock>>>(d_in, d_out, length, width);
    hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());

    checkCudaErrors(hipMemcpy(h_out, d_out, sizeof(float)*length*width, hipMemcpyDeviceToHost));
    
    checkCudaErrors(hipFree(d_out));
    checkCudaErrors(hipFree(d_in));
    return;
}
void float2cuComplex(float *h_in, hipComplex *h_out, const int length, const int width)
{
    float *d_in;
    hipComplex *d_out;

    checkCudaErrors(hipMalloc((void**)&d_in, sizeof(float)*length*width));
    checkCudaErrors(hipMalloc((void**)&d_out, sizeof(hipComplex)*length*width));
        
    checkCudaErrors(hipMemcpy(d_in, h_in, sizeof(float)*length*width, hipMemcpyHostToDevice));

    dim3 threadsPerBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 numOfBlocks(length/threadsPerBlock.x + 1, width/threadsPerBlock.y + 1);

    float2cuComplex_kernel<<<numOfBlocks, threadsPerBlock>>>(d_in, d_out, length, width);
    hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());

    checkCudaErrors(hipMemcpy(h_out, d_out, sizeof(hipComplex)*length*width, hipMemcpyDeviceToHost));
    
    checkCudaErrors(hipFree(d_in));
    checkCudaErrors(hipFree(d_out));
    return;
}
void make_gridValues(hipComplex *gridValues, const double kxMin, const int mapWidth, const int nInterpSidelobes, const double dkx)
{
    hipComplex *d_gridValues;

    checkCudaErrors(hipMalloc((void**)&d_gridValues, sizeof(hipComplex)*mapWidth));
    
    dim3 threadsPerBlock(BLOCK_SIZE);
    dim3 numOfBlocks(mapWidth/threadsPerBlock.x + 1);

    make_gridValues_kernel<<<numOfBlocks, threadsPerBlock>>>(d_gridValues, kxMin, mapWidth, nInterpSidelobes, dkx);
    hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());

    checkCudaErrors(hipMemcpy(gridValues, d_gridValues, sizeof(hipComplex)*mapWidth, hipMemcpyDeviceToHost));

    checkCudaErrors(hipFree(d_gridValues));
    return;
}
void zero_mat(hipComplex *h_mat, const int length, const int width)
{
    hipComplex *d_mat;

    checkCudaErrors(hipMalloc((void**)&d_mat, sizeof(hipComplex)*length*width));
    checkCudaErrors(hipMemcpy(d_mat,   h_mat, sizeof(hipComplex)*length*width, hipMemcpyHostToDevice));

    dim3 threadsPerBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 numOfBlocks(length/threadsPerBlock.x + 1, width/threadsPerBlock.y + 1);

    zero_mat_kernel<<<numOfBlocks, threadsPerBlock>>>(d_mat, length, width);
    hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());

    checkCudaErrors(hipMemcpy(h_mat, d_mat, sizeof(hipComplex)*length*width, hipMemcpyDeviceToHost));
    checkCudaErrors(hipFree(d_mat));
    return;
}

void print_mat(hipComplex *mat, const int xVal, const int yVal)
{
    hipComplex curr;
    for(int x = 0; x < xVal; x++)
    {
        for(int y = 0; y < yVal; y++)
        {
            curr = mat[x*yVal + y];
            printf( "%g + (%gi), ", hipCrealf(curr), hipCimagf(curr));
        }
        cout << endl;
    }
}

void print_mat_file(char* filename, hipComplex *mat, const int xVal, const int yVal)
{
    hipComplex curr;
    FILE * myFile;
    myFile = fopen (filename, "w");
    for(int x = 0; x < xVal; x++)
    {
        for(int y = 0; y < yVal; y++)
        {
            curr = mat[x*yVal + y];
            fprintf(myFile, "%g + (%gi), ", hipCrealf(curr), hipCimagf(curr));
        }
        fprintf(myFile, "\n");
    }
    fclose (myFile);
}

// Produces Compression Constants
void comp_decomp(const float Xc, hipComplex *uc, const int length,  hipComplex *u, const int u_len, hipComplex *k, const int width, hipComplex *compression, hipComplex *decompression)
{
    hipComplex *f_uc, *f_u, *f_k;

    f_u  = (hipComplex *)malloc(sizeof(hipComplex)*u_len);
    f_uc = (hipComplex *)malloc(sizeof(hipComplex)*length);
    f_k  = (hipComplex *)malloc(sizeof(hipComplex)*width);

    // fftshift
    fftshift(uc, f_uc, length, 1, 0);
    fftshift(u, f_u, u_len, 1, 0);

    // Square each element
    square(f_uc, f_uc, length, 1);
    square(f_u, f_u, u_len, 1);

    // add constant to vector
    sca_vec_add(Xc*Xc, f_uc, length, 1, 1);
    sca_vec_add(Xc*Xc, f_u, u_len, 1, 1);

    // sqrt(abs complex vector)
    sqrt_abs(f_uc, f_uc, length, 1);
    sqrt_abs(f_u, f_u, u_len, 1);

    // subtract contant from vector
    sca_vec_add(-1.0*Xc, f_uc, length, 1, 1.0);
    // Xc - u
    sca_vec_add(Xc, f_u, u_len, 1, -1.0);

    // change real vector imaginary vector
    real_to_imag(k, f_k, width, 1);
    sca_vec_mult(2.0, f_k, width, 1);

    // mult vec vec to matrix
    vec_vec_mat(f_uc, f_k, compression, length, width);
    vec_vec_mat(f_k, f_u, decompression, width, u_len);

    // exp mat
    exp_mat(compression, compression, width, length);
    exp_mat(decompression, decompression, u_len, width);

    free(f_u);
    free(f_uc);
    free(f_k);
    return;
}

void fft(hipComplex *h_matrix, hipComplex *h_out, const int length, const int width, int direction)
{   // One dimensional fft along length
    hipComplex *d_matrix, *d_out;
    hipfftHandle plan;

    checkCudaErrors(hipMalloc((void**)&d_matrix, sizeof(hipComplex)*length*width));
    checkCudaErrors(hipMalloc((void**)&d_out, sizeof(hipComplex)*length*width));
    
    checkCudaErrors(hipMemcpy(d_matrix, h_matrix, sizeof(hipComplex)*length*width, hipMemcpyHostToDevice));

    hipfftPlan1d(&plan, length, HIPFFT_C2C, width);

    hipfftExecC2C(plan, d_matrix, d_out, direction);

    checkCudaErrors(hipMemcpy(h_out, d_out, sizeof(hipComplex)*length*width, hipMemcpyDeviceToHost));

    hipfftDestroy(plan);
    checkCudaErrors(hipFree(d_out));
    checkCudaErrors(hipFree(d_matrix));
    return;
}

void fast_time_block(hipComplex *sRaw, hipComplex *fast_time_filter, const int length, const int width, 
                     const int mapWidth, hipComplex *fsSpotLit)
{
    hipComplex *fast_time_filter_copy, *sRaw_copy, *compression, *decompression, *uc, *u, *k;
    float Xc = 1000.0;

    sRaw_copy = (hipComplex *)malloc(sizeof(hipComplex)*length*width);
    fast_time_filter_copy = (hipComplex *)malloc(sizeof(hipComplex)*length);

    fft(sRaw, sRaw_copy, length, width, HIPFFT_FORWARD);
    fftshift(sRaw_copy, sRaw_copy, length, width, 2);
    //sraw

    fftshift(fast_time_filter, fast_time_filter_copy, length, 1, 0);
    mat_vec_mult(sRaw_copy, fast_time_filter_copy, sRaw_copy, length, width);
    free(fast_time_filter_copy);
    
    uc = (hipComplex *)malloc(sizeof(hipComplex)*width);
    k  = (hipComplex *)malloc(sizeof(hipComplex)*length);
    u  = (hipComplex *)malloc(sizeof(hipComplex)*mapWidth);
    compression = (hipComplex *)malloc(sizeof(hipComplex)*length*width);
    decompression = (hipComplex *)malloc(sizeof(hipComplex)*length*mapWidth);
    
    csv_real_reader("u.csv",   u, true, true);
    csv_real_reader("uc.csv", uc, true, true);
    csv_real_reader("k.csv",   k, true, true);
    comp_decomp(Xc, uc, width, u, mapWidth, k, length, compression, decompression);

    free(u);
    free(uc);

    vec_vec_mult(sRaw_copy, compression, sRaw_copy, length, width);

    free(compression);
    
    transpose(sRaw_copy, length, width);
    
    fft(sRaw_copy, sRaw_copy, width, length, HIPFFT_FORWARD);
    
    pad(sRaw_copy, fsSpotLit, width, length, width/2, mapWidth - width);
    
    sca_vec_mult(382.0/160.0, fsSpotLit, mapWidth, length);

    fft(fsSpotLit, fsSpotLit, mapWidth, length, HIPFFT_BACKWARD);
    sca_vec_mult(1.0/mapWidth, fsSpotLit, mapWidth, length);

    vec_vec_mult(fsSpotLit, decompression, fsSpotLit, mapWidth, length);
    
    free(decompression);
    
    fft(fsSpotLit, fsSpotLit, mapWidth, length, HIPFFT_FORWARD);

    fftshift(fsSpotLit, fsSpotLit, mapWidth, length, 2);
    
    transpose(fsSpotLit, mapWidth, length);

    free(sRaw_copy);
    return;
}

float sinc(float x)
{
    if (x == 0)
        return 1.0;
    else
        return sin(x*PI)/(PI*x);
}

void SpatialInterpolate(hipComplex *filteredSignal, float *wn, hipComplex *GridValues,
                        float dkx, float kxs, const int length, const int mapWidth, const int mapLength, 
                        hipComplex *outSignal, hipComplex *idxout)
{
    const int nInterpSidelobes = 8;
    
    hipComplex *row;
    hipComplex *d_filteredSignal, *d_outSignal, *d_rowidx, *d_slicerange;
    int *rowidx;

    float *d_wn, *d_gridvalues, *gridvalues;

    row = (hipComplex *)malloc(sizeof(hipComplex)*length*mapWidth);
    rowidx = (int *)malloc(sizeof(int)*length*mapWidth);
    gridvalues = (float *)malloc(sizeof(float)*mapLength);
    
    float2cuComplex(wn, row, length, mapWidth);

    float negGridValue = -1.0*hipCabsf(GridValues[0]);

    sca_vec_add(negGridValue, row, length, mapWidth, 1.0);

    sca_vec_mult(1.0/dkx, row, length, mapWidth);

    round_vec(row, idxout, length, mapWidth);

    sca_vec_add(-1.0*nInterpSidelobes, idxout, length, mapWidth, 1.0);

    cuComplex2Int(idxout, rowidx, length, mapWidth);

    cuComplex2float(GridValues, gridvalues, 1, mapLength);
    //Everything works before this line

    //rowidx is complete at this line
    
    //MAKE IDXOUT
    //MAKE SLICERANGE
    //MAKE HAM*SNC

    //Setting up Cuda memory
    checkCudaErrors(hipMalloc((void**)&d_wn,                    sizeof(float)*length*mapWidth));
    checkCudaErrors(hipMalloc((void**)&d_rowidx,                sizeof(hipComplex)*length*mapWidth));
    checkCudaErrors(hipMalloc((void**)&d_gridvalues,            sizeof(float)*mapLength));
    checkCudaErrors(hipMalloc((void**)&d_filteredSignal,        sizeof(hipComplex)*length*mapWidth));
    checkCudaErrors(hipMalloc((void**)&d_slicerange,        sizeof(hipComplex)*length*mapWidth));
    checkCudaErrors(hipMalloc((void**)&d_outSignal,             sizeof(hipComplex)*mapLength*mapWidth));

    checkCudaErrors(hipMemcpy(d_wn, wn,                         sizeof(float)*mapWidth*length,        hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_rowidx, idxout,                 sizeof(hipComplex)*mapWidth*length,          hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_gridvalues, gridvalues,         sizeof(float)*mapLength,              hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_filteredSignal, filteredSignal, sizeof(hipComplex)*mapWidth*length,    hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_outSignal, outSignal,           sizeof(hipComplex)*mapWidth*mapLength, hipMemcpyHostToDevice));
    
    dim3 threadsPerBlock(BLOCK_SIZE, BLOCK_SIZE, 1);
    dim3 numOfBlocks(length/threadsPerBlock.x + 1, mapWidth/threadsPerBlock.y + 1, 2*nInterpSidelobes);
    //->
    spatial_inter_kernel<<<numOfBlocks, threadsPerBlock>>>(d_filteredSignal, d_wn,
        d_gridvalues, d_rowidx, length, mapLength, mapWidth, nInterpSidelobes, dkx, kxs, 
        d_outSignal, d_slicerange);
    hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());

    checkCudaErrors(hipMemcpy(outSignal, d_outSignal, sizeof(hipComplex)*mapWidth*mapLength, hipMemcpyDeviceToHost));
    checkCudaErrors(hipMemcpy(idxout, d_slicerange, sizeof(hipComplex)*length*mapWidth, hipMemcpyDeviceToHost));

    free(row);
    free(rowidx);
    checkCudaErrors(hipFree(d_wn));
    checkCudaErrors(hipFree(d_rowidx));
    checkCudaErrors(hipFree(d_gridvalues));
    checkCudaErrors(hipFree(d_outSignal));
    checkCudaErrors(hipFree(d_filteredSignal));
    return;
}

void SpatialInterpolate_seq(hipComplex *filteredSignal, float *wn, hipComplex *GridValues,
                        float dkx, float kxs, const int length, const int mapWidth, const int mapLength, 
                        hipComplex *outSignal, hipComplex *idxout)
{
     const int nInterpSidelobes = 8;
    
    hipComplex *row;
    int *rowidx;

    row = (hipComplex *)malloc(sizeof(hipComplex)*length*mapWidth);
    rowidx = (int *)malloc(sizeof(int)*length*mapWidth);
    
    float2cuComplex(wn, row, length, mapWidth);

    float negGridValue = -1.0*hipCabsf(GridValues[0]);

    sca_vec_add(negGridValue, row, length, mapWidth, 1.0);

    sca_vec_mult(1.0/dkx, row, length, mapWidth);

    round_vec(row, idxout, length, mapWidth);

    sca_vec_add(-1.0*nInterpSidelobes, idxout, length, mapWidth, 1.0);

    cuComplex2Int(idxout, rowidx, length, mapWidth);
    int ix,iy, ind;
    float slicerange;
    long int idx, imag_ind;
    float ham;
    for(int iz = 0; iz < 15; iz++)
    {
        for(iy = 0; iy < mapWidth; iy++)
        {
            for(ix = 0; ix < length; ix++)
            {
                ind = iy*length+ix;
                idx = rowidx[iy*length+ix] + iz;
                slicerange = GridValues[idx].x - wn[ind];
                ham = (0.54 + 0.46*cos( slicerange *(PI/kxs)))*sinc(slicerange/dkx);
                imag_ind = idx + mapLength*iy;
                outSignal[imag_ind].x = outSignal[imag_ind].x + filteredSignal[ind].x*ham;
                outSignal[imag_ind].y = outSignal[imag_ind].y + filteredSignal[ind].y*ham;
            }
        }
    }

    return;
}

int main()
{
	//new code
	hipDeviceReset();
	hipSetDevice(0);

	// Get all data
	ifstream fastTimeFilter ("fastTimeFilter.csv");
	// End of data read

	float d, i;
    float m = 382;
    float mc = 160;
    double Xc = 1000.0;
    double dkx = 0.0785;
    double kxs = 0.6283;
    double kxMin = 5.8462;

    int nInterpSidelobes = 8;

    //Dimensions of sRaw data
    int width = 438;
    int batch = 160;

    //Dimensions of final map
    int mapLength = 382;
    int mapWidth  = 266;

	hipComplex *sRaw, *signal, *mapOut, *out_signal, *u, *uc, *k, *ku0, *padded_data, *fsSpotLit, *sig;
    hipComplex *kmat, *ku0mat, *kx, *kx_gt_zero, *kx_work, *filteredSignal, *no_interpolation_image;
    hipComplex *finalImage, *idxout, *gridValues;
    float *kx_float;
    
	u           = (hipComplex *)malloc(sizeof(hipComplex)*mapLength);
    uc          = (hipComplex *)malloc(sizeof(hipComplex)*batch);
    k           = (hipComplex *)malloc(sizeof(hipComplex)*width);
    ku0         = (hipComplex *)malloc(sizeof(hipComplex)*mapLength);
    signal      = (hipComplex *)malloc(sizeof(hipComplex)*width);
	sRaw        = (hipComplex *)malloc(sizeof(hipComplex)*width*batch);
    out_signal  = (hipComplex *)malloc(sizeof(hipComplex)*width*batch);
    padded_data = (hipComplex *)malloc(sizeof(hipComplex)*width*mapLength);
    fsSpotLit   = (hipComplex *)malloc(sizeof(hipComplex)*width*mapLength);
    sig         = (hipComplex *)malloc(sizeof(hipComplex)*width*mapLength);

    csv_real_reader("u.csv",   u, true, true);
    csv_real_reader("uc.csv", uc, true, true);
    csv_real_reader("k.csv",   k, true, true);
    csv_real_reader("ku0.csv", ku0, true, true);
    csv_real_reader("imagsRaw.csv", sRaw, false, false);
    csv_real_reader("realsRaw.csv", sRaw, true, false);
    csv_real_reader("imagfilteredSignal.csv", sig, false, false);
    csv_real_reader("RealfilteredSignal.csv", sig, true, false);
    
    //Copying Data from CSV files into memory
    string value;
    list<string> values;
    while ( fastTimeFilter.good() )
    {
        // read a string until next comma: http://www.cplusplus.com/reference/string/getline/
        getline ( fastTimeFilter, value, ',' ); 
        if (value.find('\n') != string::npos)
            split_line(value, "\n", values);
        else
            values.push_back(value);
    }
    //From fast time filter we get p*(-t)
    list<string>::const_iterator it = values.begin();
    int count = 0;
    for (it = values.begin(); it != values.end(); it++) {
        string tmp = *it;
        d = stof(tmp.c_str(), NULL);
		it++;
		tmp = *it;
		i = stof(tmp.c_str(), NULL);
		signal[count].x = d;
		signal[count].y = i;
		count++;
    }
	
	//Done Reading in values from files.

    //---------------------------------------------------------------------------------------------------------------------

    transpose(sRaw, batch, width);
    fast_time_block(sRaw, signal, width, batch, mapLength, fsSpotLit);
    
    // Two-D Matched Fitler
    square(k, k, width, 1);
    square(ku0, ku0, mapLength, 1);

    sca_vec_mult(4.0, k, width, 1);
    sca_vec_mult(-1.0, ku0, mapLength, 1);

    kmat   = (hipComplex *)malloc(sizeof(hipComplex)*mapLength*width);
    ku0mat = (hipComplex *)malloc(sizeof(hipComplex)*mapLength*width);
    kx     = (hipComplex *)malloc(sizeof(hipComplex)*mapLength*width);
    kx_work= (hipComplex *)malloc(sizeof(hipComplex)*mapLength*width);
    kx_gt_zero = (hipComplex *)malloc(sizeof(hipComplex)*mapLength*width);
    filteredSignal = (hipComplex *)malloc(sizeof(hipComplex)*mapLength*width);
    no_interpolation_image = (hipComplex *)malloc(sizeof(hipComplex)*mapLength*width);

    vec_copy2mat(k, kmat, width, mapLength);
    vec_copy2mat(ku0, ku0mat, mapLength, width);

    transpose(ku0mat, mapLength, width);

    vec_vec_add(kmat, ku0mat, kx, width, mapLength);
    sca_max(0, kx, kx, width, mapLength);

    sqrt_abs(kx, kx, width, mapLength);
    // kx is kx at this point

    is_pos(kx, kx_gt_zero, width, mapLength);

    sqrt_abs(kmat, kmat, width, mapLength);

    sca_vec_mult(-1.0, kmat, width, mapLength);

    vec_vec_add(kx, kmat, kx_work, width, mapLength);
    
    sca_vec_mult(Xc, kx_work, width, mapLength);

    csv_real_reader("ku0.csv", ku0, true, true);

    fftshift(ku0, ku0, mapLength, 1, 0);

    vec_copy2mat(ku0, ku0mat, mapLength, width);

    transpose(ku0mat, mapLength, width);
    
    vec_vec_add(kx_work, ku0mat, kx_work, width, mapLength);

    sca_vec_add(0.25*PI, kx_work, width, mapLength, 1.0);

    real_to_imag(kx_work, kx_work, width, mapLength);

    exp_mat(kx_work, kx_work, width, mapLength);

    vec_vec_mult(kx_work, kx_gt_zero, kx_work, width, mapLength);

    vec_vec_mult(kx_work, fsSpotLit, filteredSignal, width, mapLength);

    fft(filteredSignal, no_interpolation_image, width, mapLength, HIPFFT_BACKWARD);
    transpose(no_interpolation_image, width, mapLength);
    fft(no_interpolation_image, no_interpolation_image, mapLength, width, HIPFFT_BACKWARD);
    transpose(no_interpolation_image, mapLength, width);

    sca_vec_mult(1.0/(mapLength*sqrt(2)*width), no_interpolation_image, width, mapLength);
    print_mat_file("no_interpolation.txt", no_interpolation_image, width, mapLength);

    sca_vec_mult(1.0/width, no_interpolation_image, width, mapLength);
    sca_vec_mult(1.0/mapLength, no_interpolation_image, width, mapLength);
    
    kx_float = (float *)malloc(sizeof(float)*mapLength*width);
    cuComplex2float(kx, kx_float, width, mapLength);
    
    finalImage = (hipComplex *)malloc(sizeof(hipComplex)*mapLength*mapWidth);
    
    zero_mat(finalImage, mapLength, mapWidth);
    
    gridValues = (hipComplex *)malloc(sizeof(hipComplex)*mapWidth);

    make_gridValues(gridValues, kxMin, mapWidth, nInterpSidelobes, dkx);

    // Spatial Interpolate
    idxout = (hipComplex *)malloc(sizeof(hipComplex)*width*mapLength);
    SpatialInterpolate(filteredSignal, kx_float, gridValues, dkx, kxs, width, mapLength, mapWidth, finalImage, idxout);
    
    // ifft2
    fft(finalImage, finalImage, mapWidth, mapLength, HIPFFT_BACKWARD);
    transpose(finalImage, mapWidth, mapLength);
    fft(finalImage, finalImage, mapLength, mapWidth, HIPFFT_BACKWARD);
    transpose(finalImage, mapLength, mapWidth);
    
    sca_vec_mult(1.0/(mapLength*sqrt(2)*mapWidth), finalImage, mapWidth, mapLength);

    print_mat_file("finalImage.txt", finalImage, mapLength, mapWidth);

    free(u);
    free(k);
    free(uc);
    free(ku0);
    free(kmat);
    free(sRaw);
    free(ku0mat);
    free(idxout);
    free(signal);
    free(kx_work);
    free(kx_float);
    free(fsSpotLit);
    free(out_signal);
    free(finalImage);
    free(padded_data);    
    free(no_interpolation_image);
	return 0;
}
